/*
 * Compile:
 *
 * nvcc -g -arch=sm_30 nvcc_type_sizes.cu -o nvcc_type_sizes
 *
 */

#include <stdio.h>
#include <hip/hip_runtime.h>



void cudaAssert(const hipError_t err, const char *file, const int line)
{ 
    if( hipSuccess != err) {                                                
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        
                file, line, hipGetErrorString(err) );
        exit(1);
    } 
}

__global__ void print_sizes() {

  printf("sizeof(int) = \t\t%ld bytes = %ld bits\n", sizeof(int), 
	 8*sizeof(int)); 
  printf("sizeof(long) = \t\t%ld bytes = %ld bits\n", sizeof(long), 
	 8*sizeof(long)); 
  printf("sizeof(long long) = \t%ld bytes = %ld bits\n", sizeof(long long), 
	 8*sizeof(long long)); 
  printf("sizeof(float) = \t%ld bytes = %ld bits\n", sizeof(float), 
	 8*sizeof(float)); 
  printf("sizeof(double) = \t%ld bytes = %ld bits\n", sizeof(double), 
	 8*sizeof(double)); 
  printf("sizeof(long double) = \t%ld bytes = %ld bits\n", sizeof(long double), 
	 8*sizeof(long double)); 

}

int main() {
    print_sizes<<<1,1>>>();
    hipDeviceSynchronize();
  return 0;
}
