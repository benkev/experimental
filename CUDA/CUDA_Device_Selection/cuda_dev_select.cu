//
// Compile:
// $ nvcc cuda_dev_select.cu -o cuda_dev_select
//
//

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>


int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    int device;
    for (device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("Device %d, %s, compute capability %d.%d\n",
               device, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    // printf("\n\n");
    printf("deviceCount = %d\n", deviceCount);
    
}
